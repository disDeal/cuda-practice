
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define N (1024*1024)

__global__ void kernel(float* dA) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float x = 2.0f * 3.1415926f * (float)idx / (float)N;
  dA[idx] = sinf(sqrtf(x));
}

int main(int argc, char* argv[]) {
  float* hA; float* dA;
  hA = (float*) malloc(N * sizeof(float));
  hipMalloc((void**) &dA, N*sizeof(float));
  kernel<<<N/512, 512>>>(dA);
  hipMemcpy(hA, dA, N*sizeof(float), hipMemcpyDeviceToHost);
  for (int idx = 0; idx < N; idx++) {
    printf("a[%d] = %0.5f\n", idx, hA[idx]);
  } 
  free(hA);
  hipFree(dA);
  return 0;
} 